#include "hip/hip_runtime.h"
// Matrix transpose using shared memory to ensure all writes coalesce.
// Example for video 4.1.

#include <assert.h>
#include <memory>

// Standard CUDA API functions
#include <hip/hip_runtime_api.h>

// CUDA cooperative groups API
#include <hip/hip_cooperative_groups.h>

#include "../utils.h"

const int TILE_DIM = 16;

// Reference implementation on the host
void transpose_reference(const float *source, float *dest,
                         unsigned int dimension)
{
  for (int y = 0; y < dimension; y++) {
    for (int x = 0; x < dimension; x++) {
      dest[y + x * dimension] = source[x + y * dimension];
    }
  }
}

// Transpose a matrix
// For simplicity, we assume that the matrix is square, and that its
// dimension is a multiple of the block size, so we don't have to worry about
// pitch or bounds checking.
__global__ void transpose(const float *source, float *dest,
                          unsigned int dimension)
{
  // Shared memory to temporarily store data.
  // Note the padding of the Y dimension, to avoid bank conflicts.
  __shared__ float tile[TILE_DIM][TILE_DIM + 1];

  int x_in = blockIdx.x * blockDim.x + threadIdx.x;
  int y_in = blockIdx.y * blockDim.y + threadIdx.y;
  int source_index = x_in + y_in * dimension;

  // Read from global memory to shared memory. Global memory access is
  // aligned.
  tile[threadIdx.y][threadIdx.x] = source[source_index];

  // Wait for all threads in the block to finish, so the shared memory tile
  // is filled.
  cooperative_groups::thread_block block =
      cooperative_groups::this_thread_block();
  cooperative_groups::sync(block);

  // Output coordinates. Note that blockIdx.y is used to determine x_out, and
  // blockIdx.x is used to determine y_out.
  int x_out = blockIdx.y * blockDim.y + threadIdx.x;
  int y_out = blockIdx.x * blockDim.y + threadIdx.y;
  int dest_index = x_out + y_out * dimension;

  // Read from a different index in the shared memory tile, and write to
  // global memory. Global memory access is once again aligned.
  dest[dest_index] = tile[threadIdx.x][threadIdx.y];
}

int main(int argc, char **argv)
{
  const unsigned int DIMENSION = 4096;
  const unsigned int COUNT = DIMENSION * DIMENSION;
  std::unique_ptr<float[]> source(new float[COUNT]);
  std::unique_ptr<float[]> dest(new float[COUNT]);

  // Fill source matrix with some arbitrary test values
  for (int i = 0; i < COUNT; i++) {
    source[i] = i;
  }

  // Allocate and fill device memory
  float *source_dev, *dest_dev;
  size_t size = COUNT * sizeof(float);
  cudaCheckError(hipMalloc(&dest_dev, size));
  cudaCheckError(hipMalloc(&source_dev, size));
  cudaCheckError(
      hipMemcpy(source_dev, source.get(), size, hipMemcpyHostToDevice));

  // Run the kernel
  dim3 block_dim(TILE_DIM, TILE_DIM);
  dim3 grid_dim((DIMENSION + block_dim.x - 1) / block_dim.x,
                (DIMENSION + block_dim.y - 1) / block_dim.y);

  {
    KernelTimer t;
    transpose<<<grid_dim, block_dim>>>(source_dev, dest_dev, DIMENSION);
  }

  // Copy results back to the host
  cudaCheckError(
      hipMemcpy(dest.get(), dest_dev, size, hipMemcpyDeviceToHost));
  cudaCheckError(hipFree(dest_dev));
  cudaCheckError(hipFree(source_dev));

  // Compare with reference implementation
  std::unique_ptr<float[]> dest_reference(new float[COUNT]);
  transpose_reference(source.get(), dest_reference.get(), DIMENSION);

  for (int i = 0; i < COUNT; i++) {
    assert(dest_reference.get()[i] == dest.get()[i]);
  }

  return 0;
}
