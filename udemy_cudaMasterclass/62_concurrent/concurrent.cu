#include <cstdio>
#include <cstdlib>
#include "hip/hip_runtime.h"

#include <iostream>
using std::cout;
using std::endl;
__global__ void simple_kernel() {
  printf("Hello from the kernel\n");
}

int main(int argc, char** argv) {
  int dev = 0;
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, dev);
  if (deviceProp.concurrentKernels == 0)
    cout <<"> GPU does not support concurrent kernel execution\n";

  hipStream_t str1, str2, str3;
  hipStreamCreate(&str1);
  hipStreamCreate(&str2);
  hipStreamCreate(&str3);

  simple_kernel <<< 1,1,0,str1 >>>();
  simple_kernel <<< 1,1,0,str2 >>>();
  simple_kernel <<< 1,1,0,str3 >>>();

  hipStreamDestroy(str1);
  hipStreamDestroy(str2);
  hipStreamDestroy(str3);
  hipDeviceSynchronize();
  hipDeviceReset();
  return 0;
}
